
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <string.h>
#include <fstream>
#include <sstream>
//#include <bits/stdc++.h>
//#include <stdlib.h>
//#include <time.h>
using namespace std;
/***DEFINING THE DEFINES FOR THE ARRAY INDICES****************************/
//#define N 128
#define C 3
#define H 227
#define W 227
#define R 11
#define S 11
#define M 96
#define E 55
#define F 55
#define U 4
__global__
void ew_gpu_mmul(float* d_o, float* d_i, float* d_w, int width, int height, int stride, int ip_height, int wt_width, int num_wt,int num_img, int num_ch)
{
int row = threadIdx.y; int col = threadIdx.x;
{
for(int x=0;x<2;x++){
for(int y=0;y<2;y++){
for (int i=0; i<wt_width; i++){
  for (int j=0; j<wt_width; j++){
   for(int k=0; k<num_ch; k++){
//    float ip = d_i[blockIdx.x*num_ch*ip_height*ip_height+k*ip_height*ip_height+(stride*(2*row+y)+i)*ip_height+(stride*(2*col+x)+j)];
  //      float wt = d_w[blockIdx.y*num_ch*wt_width*wt_width+k*wt_width*wt_width+(i*wt_width+j)];
    //    float  prod = ip*wt;
        if((2*row+y<height)&&(2*col+x<width))
         d_o[blockIdx.x*num_wt*height*width+blockIdx.y*width*height+(2*row+y)*width+(2*col+x)] +=d_i[blockIdx.x*num_ch*ip_height*ip_height+k*ip_height*ip_height+(stride*(2*row+y)+i)*ip_height+(stride*(2*col+x)+j)]*d_w[blockIdx.y*num_ch*wt_width*wt_width+k*wt_width*wt_width+(i*wt_width+j)] ;

}
}
}
//if(d_o[blockIdx.x*num_wt*height*width+blockIdx.y*width*height+(2*row+y)*width+(2*col+x)]<0)
//d_o[blockIdx.x*num_wt*height*width+blockIdx.y*width*height+(2*row+y)*width+(2*col+x)] =0;
}
}
}
}

 

void element_wise_mmul(float* output, float* input, float* weight, int batch_size)
{
int x,y,i,j,m,n,k;
for(n=0; n<batch_size; n++){
for (m=0 ; m<M; m++){
 for (x=0; x<F; x++){
         for(y=0; y<E; y++){
            //    OP[x][y] = 0; // adding bias to output
                 for (i=0; i<R; i++){
                         for (j=0; j<S; j++){
				for(k=0; k<C; k++){
				float ip = input[n*C*H*W+k*H*W+(U*x+i)*H+(U*y+j)];
				float wt = weight[m*C*R*S+k*R*S+i*S+j];

				float prod = ip*wt;
				if(prod >=0)
				output[n*E*F*M+m*E*F+x*E+y] += prod;
                                 //OP[x][y] += IP[U*x+i][U*y+j]*WT[i][j];
                                                         }}
                                                 }
                                        }
                                         }
 

}
}
}
int main(int argc, char* argv[])
{
int batch_size = atoi(argv[1]);
/*************INITALIZING MATRICES*********************************/
float* IP;
//float *IP = (float*) malloc(batch_size*C*H*W*sizeof(float));
hipMallocManaged(&IP,batch_size*C*H*W*sizeof(float));
//float IP[H][W];
float *OP = (float*) malloc(batch_size*M*F*E*sizeof(float));
//float OP[F][E];
float* OPG; 
//float *OPG = (float*) malloc(batch_size*M*F*E*sizeof(float));
hipMallocManaged(&OPG,batch_size*M*F*E*sizeof(float));

float* WT;
//float *WT = (float*) malloc(M*C*R*S*sizeof(float));
hipMallocManaged(&WT,M*R*C*S*sizeof(float));
//float WT[R][S];
//float* d_o;
//float* d_i;
//float* d_w;

//clock_t cpu_start, gpu_start, cpu_end, gpu_end;
//int a,b,c,d;
int c,d,m,n,k;
/*INITIALIZING WEIGHT MATRIX*/
for (m=0; m<M; m++){
for(k=0;k<C;k++){
for (c=0; c<R; c++){
	for(d=0; d<S; d++){
		//WT[c][d] = 2.0;
		WT[m*C*R*S+k*R*S+c*S+d] = (float)rand()/(RAND_MAX+1.0);
}
}	
}
}
/*INITIALIZING OUTPUT MATRIX*/
for (n=0; n<batch_size;n++){
for (m=0; m<M; m++){
for (c=0; c<F; c++){
	for(d=0; d<E; d++){
		//OP[c][d] = 0;
		OP[n*M*F*E+m*F*E+c*E+d] = 0;
}
}	
}
}
/*INITIALIZING INPUT MATRIX*/
for (n=0; n<batch_size; n++){
for(k=0;k<C;k++){
for (c=0; c<H; c++){
	for(d=0; d<W; d++){
	//	IP[c][d] = (a+b+c+d);
	//if ((c<=1) || (d<=1) || (c>=29) || (d>=29))
        //IP[n*C*H*W+k*H*W+c*W+d] = 0;
        //else
        IP[n*C*H*W+k*H*W+c*W+d] = (float)rand()/(RAND_MAX+1.0);

//		IP[n*C*H*W+k*H*W+c*W+d] = (c+d);
}
}	
}
}
//cudaMalloc((void**) &d_i,batch_size*C*H*W*sizeof(float));
//cudaMemcpy(d_i, IP, batch_size*C*H*W*sizeof(float), cudaMemcpyHostToDevice);
//cudaMalloc((void**) &d_w, M*C*R*S*sizeof(float));
//cudaMemcpy(d_w, WT, M*C*R*S*sizeof(float), cudaMemcpyHostToDevice);
//cudaMalloc((void**) &d_o, batch_size*M*E*F*sizeof(float));
clock_t start, end;
start = clock();
//element_wise_mmul(OP, IP, WT, batch_size);
end = clock();

dim3 dimGrid(batch_size,96,1);
dim3 dimBlock(28,28,1);
//gpu_start = clock();
ew_gpu_mmul<<<dimGrid, dimBlock>>>(OPG,IP,WT,55,55,4,227,11,96,batch_size,3);
hipDeviceSynchronize();
//gpu_end = clock();
//cudaMemcpy(OPG,d_o, batch_size*M*E*F*sizeof(float), cudaMemcpyDeviceToHost);
/***NAIVE 7 LAYER LOOP IMPLEMENTATION***/
/*
int n,m,x,y,i,j,k;
for (x=0; x<F; x++){
	for(y=0; y<E; y++){
		OP[x][y] = 0; // adding bias to output
		for (i=0; i<R; i++){
			for (j=0; j<S; j++){
				OP[x][y] += IP[U*x+i][U*y+j]*WT[i][j];
							}
						}
					}
   					}
	
*/

/**print outputs**/
//int e,f,g,h;
int g,h,s,u;
float max_error=0;
string filename = "layer_1_"+to_string(batch_size);
ifstream fin(filename.c_str());
string line ;


for (u=0;u<batch_size;u++){
for (s=0;s<M;s++){
for (g=0; g<F; g++){
	for(h=0; h<E; h++){
	getline(fin,line);
        float error = abs(OPG[u*M*F*E+s*E*F+g*E+h]-atof(line.c_str()));
	//float error = abs(OPG[u*M*F*E+s*E*F+g*E+h]-OP[u*M*F*E+s*E*F+g*E+h]);
	if(error > max_error)
	max_error = error;
//	printf("inside loop\n");	
//       printf("the output is %f for index %d, %d,%d,%d.\n",OP[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
  //     printf("diff CPU and GPU is %f for index %d,%d,%d,%d.\n", OPG[u*M*F*E+s*E*F+g*E+h]-OP[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
    //   printf("the output from GPU  is %f for index %d,%d,%d,%d.\n",OPG[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
}
}
}
}
fin.close();
printf("max error %f\n", max_error);
//cout<<"time taken by cpu call is "<<((double)(cpu_end-cpu_start))/CLOCKS_PER_SEC<<"secs"<<endl;
//cout<<"time taken by gpu call is "<<((double)(gpu_end-gpu_start))/CLOCKS_PER_SEC<<"secs"<<endl;

//cudaFree(d_o);
//cudaFree(d_i);
//cudaFree(d_w);
hipFree(OPG);
hipFree(IP);
hipFree(WT);
hipFree(OP);	
return 0;
}

