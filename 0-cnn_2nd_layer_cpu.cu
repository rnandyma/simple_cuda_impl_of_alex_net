
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <string.h>
#include <iostream>
/***DEFINING THE DEFINES FOR THE ARRAY INDICES****************************/
//#define N 1
#define C 96
#define H 31
#define W 31
#define R 5
#define S 5
#define M 256
#define E 27
#define F 27
#define U 1
using namespace std;
int main(int argc, char* argv[])
{
int batch_size = atoi(argv[1]);
/*************INITALIZING MATRICES*********************************/
float IP[batch_size][C][H][W];
float OP[batch_size][M][F][E];
float WT[M][C][R][S];
int a,b,c,d;
/*INITIALIZING WEIGHT MATRIX*/
for(a=0; a<M; a++){
	for (b=0; b<C; b++){
		for (c=0; c<R; c++){
			for(d=0; d<S; d++){
				WT[a][b][c][d] = (float)rand()/(float)(RAND_MAX+1.0);
}
}	
}
}
printf("hello after weight\n");
/*INITIALIZING OUTPUT MATRIX*/
for(a=0; a<batch_size; a++){
	for (b=0; b<M; b++){
		for (c=0; c<F; c++){
			for(d=0; d<E; d++){
				OP[a][b][c][d] = 0;
}
}	
}
}
printf("hello after op init\n");
/*INITIALIZING INPUT MATRIX*/
for(a=0; a<batch_size; a++){
	for (b=0; b<C; b++){
		for (c=0; c<H; c++){
			for(d=0; d<W; d++){
				if((c>=29)||(d>=29)||(c<=1)||(d<=1))
				IP[a][b][c][d] = 0;//zero padding to eliminate spurious values along the edges to align stride and filter boundary properly with image boundary
				else
				IP[a][b][c][d] = (float)rand()/(float)(RAND_MAX+1.0);
}
}	
}
}
printf("hello after ininting input mat\n");
/***NAIVE 7 LAYER LOOP IMPLEMENTATION***/
int n,m,x,y,i,j,k;
for(n=0; n<batch_size; n++){
	for (m=0; m<M; m++){
		for (x=0; x<F; x++){
			for(y=0; y<E; y++){
				OP[n][m][x][y] = 0; // adding bias to output
				for (i=0; i<R; i++){
					for (j=0; j<S; j++){
						for(k=0; k<C; k++){
							OP[n][m][x][y] += IP[n][k][U*x+i][U*y+j]*WT[m][k][i][j];
							}
						}
					}
                 /*****ACTIVATION FUNCTION-RELU*******/
if(OP[n][m][x][y] < 0) 
	OP[n][m][x][y] = 0;

					}
				}	
			}
		}
/**print outputs**/
string filename ="layer_2_"+to_string(batch_size);
FILE *fp=fopen(filename.c_str(),"w+");
//if(!fout)
//{printf("file handline error"); return 0;}

int e,f,g,h;
for(e=0; e<batch_size; e++){
	for (f=0; f<M; f++){
		for (g=0; g<F; g++){
			for(h=0; h<E; h++){
				//printf("the output is %f for index %d,%d,%d,%d.\n",OP[e][f][g][h], e,f,g,h);
				//string line = to_string(OP[e][f][g][h])+" "+to_string(e)+" "+to_string(f)+" "+to_string(g)+" "+to_string(h)+".";
				//fprintf(fp,"%f,%d,%d,%d,%d.\n",OP[e][f][g][h], e,f,g,h);
				fprintf(fp,"%f\n",OP[e][f][g][h]);

}
}	
}
}
fclose(fp);
return 0;
}


